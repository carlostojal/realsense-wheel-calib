#include <hip/hip_runtime.h>

// this functions implement the deprojection of a pixel to a point in the 3D space
// cuda is used to parallelize the computation and provide higher precision on the floating point operations

// TODO: call a kernel to deproject each pixel to a point
